#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * __global__ void function == "a CUDA kernel"
 * Called from the host (CPU) and executed on the device
 */
__global__ void
helloCuda()
{
    /*
     * printf is supported since "compute capability" 2.0
     * This printf is executed by all threads in the current context (specified by kernel configuration).
     * Note that the function is called from the host, but the output takes place on the host system.
     * There is an output buffer (default size 1MB) that will be flushed at the next host-device synchronization
     */
    printf("hello from thread %d\n", threadIdx.x);
    /*only the first thread in each block executes the second printf*/
    if (threadIdx.x == 0)
        printf("==========\n");
}

int
main()
{
    /* launch kernel with 1 block of 1 thread*/
    helloCuda << < 1, 1 >> > ();
    hipDeviceSynchronize();

    /* launch kernel with 1 block of 64 threads*/
    helloCuda << < 1, 64 >> > ();

    /*
     * without hipDeviceSynchronize there will be no output!
     * The program ends before flushing the printf output buffer from the last (asynchronous) kernel call
     */
    hipDeviceSynchronize();
}
