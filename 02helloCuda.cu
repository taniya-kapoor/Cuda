#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "./gpuerrchk.h"

__global__ void hello(char* arr,int *offset){
    /*threads 0,1,2,3,4 in each block do this ....*/
    if(threadIdx.x<5){
        /*... interpret content of "arr" as integer,
        perform addition with offset and write result back to "arr"*/
        arr[threadIdx.x]+=offset[threadIdx.x];
    }
}

int main(){
    int N=5;

    char a[N]="Hello";
    int b[N]={-5,16,-8,-11,-78};

    char *a_d;
    int *b_d;

    /*allocate on device*/
    hipMalloc(&a_d,N*sizeof(char));
    hipMalloc(&b_d,N*sizeof(int));

    /*print "Hello" on host*/
    printf("%s ",a);

    /*copy "Hello" array "a" and integers "b" to device
    gpuErrchk is an error checking macro defined in "./gpuerrchk"*/
    gpuErrchk( hipMemcpy(a_d,a,N*sizeof(char),hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(b_d,b,N*sizeof(int),hipMemcpyHostToDevice) );

    /*launch kernel with 1 block and 32 threads
    (transforms a_d using b_d)
    */
    hello<<<1,32>>>(a_d,b_d);
    //Error checking ....
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    //copy transformed array 'a' back to host
    gpuErrchk( hipMemcpy(a,a_d,N,hipMemcpyDeviceToHost) );

    //print "Cuda!" on host
    printf("%s\n",a);

    //free on device
    hipFree(a_d);
    hipFree(b_d);
}
