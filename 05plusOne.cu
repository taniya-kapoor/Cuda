
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <cuda_runtime.h>

__global__ void plusOne(int *a,int nbelem)
{
	if(threadIdx.x < nbelem)
		a[threadIdx.x]++;
}

int main(){
	int N=100;

	int *a;
	int *a_d;
	
	a=(int*)malloc(N*sizeof(int));
	
	hipMalloc(&a_d,N*sizeof(int));

	for(int i=0;i<N;i++){
		a[i]=i;
	}	

	hipMemcpy(a_d,a,N*sizeof(int),hipMemcpyHostToDevice);
	
	plusOne<<<1,128>>>(a_d,N);
	
	hipMemcpy(a,a_d,N*sizeof(int),hipMemcpyDeviceToHost);

	bool ok=true;
	for(int i=0;i<N;i++){
		if(a[i] != i+1)ok=false;
	}	
	printf("%s\n",ok?"ok":"not ok");
	
	free(a);
	hipFree(a_d);
}
