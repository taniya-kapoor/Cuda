
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 5

__global__ void hello(char* arr,int *offset){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;

    arr[tid]+=offset[tid];
//    arr[blockIdx.x]+=offset[blockIdx.x];
}

int main(){
    char *A;
    int *B;

    hipMallocManaged(&A,N*sizeof(char));
    hipMallocManaged(&B,N*sizeof(int));

    A[0]='H'; A[1]='e'; A[2]='l'; A[3]='l'; A[4]='o';
    B[0]=-5;  B[1]=16;  B[2]=-8;  B[3]=-11; B[4]=-78;

    printf("%s ",A);
    hello<<<2,3>>>(A,B);
    hipDeviceSynchronize();

    printf("%s\n",A);

    hipFree(A);hipFree(B);
}
