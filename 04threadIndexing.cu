
#include <hip/hip_runtime.h>
#include <stdio.h>

//return global thread ID (for 2D grids and blocks) 
__device__ int globalID_2D(){
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	return blockId * (blockDim.x * blockDim.y) + \
	 	(threadIdx.x + threadIdx.y*blockDim.x);
}

__global__ void print_ids(){
    printf("blockIdx.(x,y,z): %d %d %d\t threadIdx.(x,y,z): %d %d %d =>\t globalID: %d\n",\
                blockIdx.x,blockIdx.y,blockIdx.z, \
                threadIdx.x,threadIdx.y,threadIdx.z,globalID_2D());
    if(globalID_2D() == 0)
    	printf("==============\n");
}

int main(){
    //1D grid of 1D-blocks (1 x 16 threads )
    print_ids<<<1,16>>>();
    hipDeviceSynchronize();

	//1D grid of 1D-blocks (4 x 4 threads)
    print_ids<<<4,4>>>();
    hipDeviceSynchronize();

	//2D grid of 2D-blocks ((2x2) x (2x2) threads) 
    dim3 blocks(2,2,1);
    dim3 threads(2,2,1);
    print_ids<<<blocks,threads>>>();
    hipDeviceSynchronize();
}
