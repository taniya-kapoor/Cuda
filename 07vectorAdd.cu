#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "./gpuerrchk.h"

#define BLOCKSIZE 128

template<typename T>
__global__ void add(T* C,const T* A,const T* B,const int N){
    int tid=blockIdx.x*blockDim.x+threadIdx.x;

    if(tid<N){
        C[tid]=A[tid]+B[tid];
    }
}

int main(){
    int N=1<<20;

    int *A,*B,*C;
    int *A_d,*B_d,*C_d;

    A=(int*)malloc(N*sizeof(int));
    B=(int*)malloc(N*sizeof(int));
    C=(int*)malloc(N*sizeof(int));
    for(int i=0;i<N;i++){
        A[i]=i-N/2;
        B[i]=N/2-i;
        C[i]=i;
    }

    hipMalloc(&A_d,N*sizeof(int));
    hipMalloc(&B_d,N*sizeof(int));
    hipMalloc(&C_d,N*sizeof(int));

    hipMemcpy(A_d,A,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(B_d,B,N*sizeof(int),hipMemcpyHostToDevice);

    dim3 nbblocks((N+BLOCKSIZE-1)/BLOCKSIZE);

    add<<<nbblocks,BLOCKSIZE>>>(C_d,A_d,B_d,N);

    hipMemcpy(C,C_d,N*sizeof(int),hipMemcpyDeviceToHost);

    bool ok=true;
    for(int i=0;i<N;i++)
        if(C[i]!=0)ok=false;

    printf("%s\n",ok?"OK!":"not OK!");

    free(A);free(B);free(C);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}
