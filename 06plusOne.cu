#include "hip/hip_runtime.h"
#include <stdio.h>


#include "gpuerrchk.h"

__global__ void plusOne(int *a,int nbelem)
{
	int tid=blockIdx.x*blockDim.x + threadIdx.x;

	if(tid < nbelem)
		a[tid]++;
}

int main(){
	int N=100000;

	int *a;
	int *a_d;
	
	a=(int*)malloc(N*sizeof(int));
	gpuErrchk( hipMalloc(&a_d,N*sizeof(int)) );

	for(int i=0;i<N;i++){
		a[i]=i;
	}	

	hipMemcpy(a_d,a,N*sizeof(int),hipMemcpyHostToDevice);
	
	dim3 blocks = (N+127)/128;
	
	plusOne<<<blocks,128>>>(a_d,N);
	
	hipMemcpy(a,a_d,N*sizeof(int),hipMemcpyDeviceToHost);

	bool ok=true;
	for(int i=0;i<N;i++){
		if(a[i] != i+1)ok=false;
	}	
	printf("%s\n",ok?"ok":"not ok");
	
	free(a);
	hipFree(a_d);
}
